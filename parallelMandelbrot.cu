#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA kernel: compute one pixel's supersampled Mandelbrot average
__global__ void mandelbrot_kernel(double *result, int width, int height, int max_iter, double x_min, double x_max, double y_min, double y_max, int SS){

    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= width || py >= height) return;
    double dx = (x_max - x_min) / width;
    double dy = (y_max - y_min) / height;
    double invSS2 = 1.0 / (SS * SS);
    double sum = 0.0;

    // unroll supersample loop
    #pragma unroll
    for (int sy = 0; sy < SS; ++sy) {
        #pragma unroll
        for (int sx = 0; sx < SS; ++sx) {
            double cx = x_min + (px + (sx + 0.5) * invSS2) * dx;
            double cy = y_min + (py + (sy + 0.5) * invSS2) * dy;

            double zx = 0.0;
            double zy = 0.0;
            int iter = 0;
            double zx2 = 0.0;
            double zy2 = 0.0;

            while (zx2 + zy2 <= 4.0 && iter < max_iter) {
                zy = 2.0 * zx * zy + cy;
                zx = zx2 - zy2 + cx;
                zx2 = zx * zx;
                zy2 = zy * zy;
                ++iter;
            }
            sum += iter;
        }
    }
    result[py * width + px] = sum * invSS2;
}

//host 
extern "C" double* compute_mandelbrot_cuda( int width, int height, int max_iter, double x_min, double x_max, double y_min, double y_max, int SS){
    if (width <= 0 || height <= 0 || max_iter <= 0 || SS <= 0) {
        fprintf(stderr, "Invalid parameters for compute_mandelbrot_cuda\n");
        return NULL;
    }

    size_t total = (size_t)width * height;
    size_t bytes = total * sizeof(double);
    double *h_result = (double*)malloc(bytes);
    if (!h_result) {
        perror("malloc");
        exit(EXIT_FAILURE);
    }

    double *d_result;
    hipMalloc(&d_result, bytes);

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    mandelbrot_kernel<<<grid, block>>>( d_result, width, height, max_iter, x_min, x_max, y_min, y_max, SS);
    hipDeviceSynchronize();
    hipMemcpy(h_result, d_result, bytes, hipMemcpyDeviceToHost);
    hipFree(d_result);
    return h_result;
}
